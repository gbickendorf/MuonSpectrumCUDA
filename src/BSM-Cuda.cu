#include "hip/hip_runtime.h"
#include "legendre_rule.h"
#include "matrixelements.h"
#include <stdio.h>
#include <vector>

__device__ double me;
__device__ double m2;
__device__ double Gf;
__device__ double g;
__device__ int orderE1;
__device__ int orderE2;
__device__ int orderct1;
__device__ int orderct2;
__device__ int orderct3;
__device__ int orderp2;
__device__ int orderp3;
__device__ double m;

struct Settings {
  double me = 0.000511;
  double m2 = 0.01;
  double Gf = 0.0000116637;
  double g = 1.0;
  int orderE1 = 32;
  int orderE2 = 32;
  int orderct1 = 32;
  int orderct2 = 32;
  int orderct3 = 32;
  int orderp2 = 32;
  int orderp3 = 32;
  double m = 0.105660;
  int done = 0;
  string filename = "	";
  bool operator==(const Settings &a) const {
    return (m2 == a.m2 && done == a.done && orderE1 == a.orderE1 &&
            orderE2 == a.orderE2 && orderct1 == a.orderct1 &&
            orderct2 == a.orderct2 && orderct3 == a.orderct3 &&
            orderp2 == a.orderp2 && orderp3 == a.orderp3);
  }
} sett;

// Calculate derivative appearing in the phasespace integral
__device__ double Deriv(double E1, double E2, double ct1, double ct2,
                        double ct3, double ph2, double ph3) {
  return 2 * (E1 + E2 - m -
              (ct2 * ct3 + cos(ph2 - ph3) * pow(1 - pow(ct2, 2), 0.5) *
                               pow(1 - pow(ct3, 2), 0.5)) *
                  pow(pow(E2, 2) - pow(m2, 2), 0.5) -
              pow(pow(E1, 2) - pow(me, 2), 0.5) *
                  (ct1 * ct3 + pow(1 - pow(ct1, 2), 0.5) *
                                   pow(1 - pow(ct3, 2), 0.5) * sin(ph3)));
}

// Solve kinetic equation for E3
__device__ double CalcE3(double E1, double E2, double ct1, double ct2,
                         double ct3, double ph2, double ph3) {
  return -(pow(E1 + E2 - m -
                   (ct2 * ct3 + cos(ph2 - ph3) * pow(1 - pow(ct2, 2), 0.5) *
                                    pow(1 - pow(ct3, 2), 0.5)) *
                       pow(pow(E2, 2) - pow(m2, 2), 0.5) -
                   pow(pow(E1, 2) - pow(me, 2), 0.5) *
                       (ct1 * ct3 + pow(1 - pow(ct1, 2), 0.5) *
                                        pow(1 - pow(ct3, 2), 0.5) * sin(ph3)),
               -1) *
           (2 * E1 * E2 - 2 * E1 * m - 2 * E2 * m + pow(m, 2) + pow(m2, 2) +
            pow(me, 2) -
            2 * pow(pow(E2, 2) - pow(m2, 2), 0.5) *
                pow(pow(E1, 2) - pow(me, 2), 0.5) *
                (ct1 * ct2 + pow(1 - pow(ct1, 2), 0.5) *
                                 pow(1 - pow(ct2, 2), 0.5) * sin(ph2)))) /
         2.;
}

// Implements a test for physicality, thus avoiding complicated region
__device__ int isPhysical(double E1, double E2, double E3) {
  if (E1 < me || E2 < 0 || E3 < 0 || E1 + E2 + E3 > m)
    return 0;
  return 1;
}

// Calculate the integrals on the GPU
__global__ void CalcRes(double *d_wE1, double *d_xE1, double *d_wE2,
                        double *d_xE2, double *d_wct1, double *d_xct1,
                        double *d_wct2, double *d_xct2, double *d_wct3,
                        double *d_xct3, double *d_wp2, double *d_xp2,
                        double *d_wp3, double *d_xp3, double *d_res) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double weight = 0.0;
  d_res[i] = 0.0;
  double E1, E2, E3, ct1, ct2, ct3, ph2, ph3;
  if (i < orderE1 * orderE2 * orderct1) {
    int iE1 = i % orderE1;
    int iCt1 = (i % (orderct1 * orderE1)) / orderE1;
    int iE2 = i / (orderE1 * orderct1);
    for (int iCt2 = 0; iCt2 < orderct2; iCt2++) {
      for (int iCt3 = 0; iCt3 < orderct3; iCt3++) {
        for (int ip2 = 0; ip2 < orderp2; ip2++) {
          for (int ip3 = 0; ip3 < orderp3; ip3++) {
            E1 = d_xE1[iE1];
            E2 = d_xE2[iE2];
            ct1 = d_xct1[iCt1];
            ct2 = d_xct2[iCt2];
            ct3 = d_xct3[iCt3];
            ph2 = d_xp2[ip2];
            ph3 = d_xp3[ip3];
            E3 = CalcE3(E1, E2, ct1, ct2, ct3, ph2, ph3);
            if (isPhysical(E1, E2, E3)) {
              weight = pow(2.0 * M_PI, -7) / 8.0 * sqrt(E1 * E1 - me * me) *
                       sqrt(E2 * E2 - m2 * m2) * E3 /
                       abs(Deriv(E1, E2, ct1, ct2, ct3, ph2, ph3)) *
                       d_wct2[iCt2] * d_wct3[iCt3] * d_wp2[ip2] * d_wp3[ip3];
              d_res[i] += weight * MatrixElements::M2ScalarToLepton(
                                       E1, E2, E3, ct1, ct2, ct3, ph2, ph3);
            }
          }
        }
      }
    }
  }
}

// Carry out all integrals on the GPU, pull back results
void IntegrateOnGPU(const char *Filename) {
  // Allocate memory for the quadrature nodes and results
  double *d_wE1, *d_xE1, *d_wE2, *d_xE2, *d_wct1, *d_xct1, *d_wct2, *d_xct2,
      *d_wct3, *d_xct3, *d_wp2, *d_xp2, *d_wp3, *d_xp3, *d_res;
  double *wE1 = (double *)malloc(sett.orderE1 * sizeof(double));
  double *xE1 = (double *)malloc(sett.orderE1 * sizeof(double));
  double *wE2 = (double *)malloc(sett.orderE2 * sizeof(double));
  double *xE2 = (double *)malloc(sett.orderE2 * sizeof(double));
  double *wct1 = (double *)malloc(sett.orderct1 * sizeof(double));
  double *xct1 = (double *)malloc(sett.orderct1 * sizeof(double));
  double *wct2 = (double *)malloc(sett.orderct2 * sizeof(double));
  double *xct2 = (double *)malloc(sett.orderct2 * sizeof(double));
  double *wct3 = (double *)malloc(sett.orderct3 * sizeof(double));
  double *xct3 = (double *)malloc(sett.orderct3 * sizeof(double));
  double *wp2 = (double *)malloc(sett.orderp2 * sizeof(double));
  double *xp2 = (double *)malloc(sett.orderp2 * sizeof(double));
  double *wp3 = (double *)malloc(sett.orderp3 * sizeof(double));
  double *xp3 = (double *)malloc(sett.orderp3 * sizeof(double));
  double *resVec = (double *)malloc(sett.orderE1 * sett.orderE2 *
                                    sett.orderct1 * sizeof(double));

  // Allocate memory on the GPU for the nodes and results
  hipMalloc(&d_wE1, sett.orderE1 * sizeof(double));
  hipMalloc(&d_xE1, sett.orderE1 * sizeof(double));
  hipMalloc(&d_wE2, sett.orderE2 * sizeof(double));
  hipMalloc(&d_xE2, sett.orderE2 * sizeof(double));
  hipMalloc(&d_wct1, sett.orderct1 * sizeof(double));
  hipMalloc(&d_xct1, sett.orderct1 * sizeof(double));
  hipMalloc(&d_wct2, sett.orderct2 * sizeof(double));
  hipMalloc(&d_xct2, sett.orderct2 * sizeof(double));
  hipMalloc(&d_xct3, sett.orderct3 * sizeof(double));
  hipMalloc(&d_wct3, sett.orderct3 * sizeof(double));
  hipMalloc(&d_wp2, sett.orderp2 * sizeof(double));
  hipMalloc(&d_xp2, sett.orderp2 * sizeof(double));
  hipMalloc(&d_wp3, sett.orderp3 * sizeof(double));
  hipMalloc(&d_xp3, sett.orderp3 * sizeof(double));
  hipMalloc(&d_res,
             sett.orderE1 * sett.orderE2 * sett.orderct1 * sizeof(double));

  // Calculate quadrature nodes
  cgqf(sett.orderct1, 1, 0, 0, -1.0, 1.0, xct1, wct1);
  cgqf(sett.orderct2, 1, 0, 0, -1.0, 1.0, xct2, wct2);
  cgqf(sett.orderct3, 1, 0, 0, -1.0, 1.0, xct3, wct3);
  cgqf(sett.orderp2, 1, 0, 0, 0.0, 2.0 * M_PI, xp2, wp2);
  cgqf(sett.orderp3, 1, 0, 0, 0.0, 2.0 * M_PI, xp3, wp3);
  cgqf(sett.orderE1, 1, 0, 0, sett.me, sett.m / 2, xE1, wE1);
  cgqf(sett.orderE2, 1, 0, 0, sett.m2, (sett.m + sett.m2) / 2.0, xE2, wE2);

  // Transfere to GPU
  hipMemcpy(d_wE1, wE1, sett.orderE1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_xE1, xE1, sett.orderE1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_wE2, wE2, sett.orderE2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_xE2, xE2, sett.orderE2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_wct1, wct1, sett.orderct1 * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_xct1, xct1, sett.orderct1 * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_wct2, wct2, sett.orderct2 * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_xct2, xct2, sett.orderct2 * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_wct3, wct3, sett.orderct3 * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_xct3, xct3, sett.orderct3 * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_wp2, wp2, sett.orderp2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_xp2, xp2, sett.orderp2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_wp3, wp3, sett.orderp3 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_xp3, xp3, sett.orderp3 * sizeof(double), hipMemcpyHostToDevice);

  // Break work into chunks and solve each on the GPU
  int N = sett.orderE1 * sett.orderE2 * sett.orderct1;
  CalcRes<<<(N + 511) / 512, 512>>>(d_wE1, d_xE1, d_wE2, d_xE2, d_wct1, d_xct1,
                                    d_wct2, d_xct2, d_wct3, d_xct3, d_wp2,
                                    d_xp2, d_wp3, d_xp3, d_res);

  // Pull back results from the GPU
  hipMemcpy(resVec, d_res, N * sizeof(double), hipMemcpyDeviceToHost);

  // Calculate the total decay width to crosscheck with montecarlo estimates
  double res = 0.0;
  for (int iE1 = 0; iE1 < sett.orderE1; iE1++) {
    for (int iCt1 = 0; iCt1 < sett.orderct1; iCt1++) {
      for (int iE2 = 0; iE2 < sett.orderE2; iE2++) {
        res += resVec[iE1 + iE2 * sett.orderE1 * sett.orderct1 +
                      iCt1 * sett.orderE1] *
               wE1[iE1] * wE2[iE2] * wct1[iCt1] / (2.0 * sett.m);
      }
    }
  }
  // Write the complete spectrum to a file
  FILE *pFile;
  pFile = fopen(Filename, "w");
  fprintf(pFile,
          "######	%.12E	%f	%f	%i	%i	%i	"
          "%i	%i	%i	%i\n",
          res, sett.m2, sett.m, sett.orderE1, sett.orderct1, sett.orderE2,
          sett.orderct2, sett.orderct3, sett.orderp2, sett.orderp3);
  for (int iE1 = 0; iE1 < sett.orderE1; iE1++) {
    for (int iCt1 = 0; iCt1 < sett.orderct1; iCt1++) {
      double buff = 0.0;
      for (int iE2 = 0; iE2 < sett.orderE2; iE2++) {
        buff += resVec[iE1 + iE2 * sett.orderE1 * sett.orderct1 +
                       iCt1 * sett.orderE1] *
                wE1[iE1] * wE2[iE2] * wct1[iCt1];
      }
      fprintf(pFile, "%.12E,%.12E,%.12E\n", xE1[iE1] * 2.0 / sett.m, xct1[iCt1],
              buff);
    }
  }
  fclose(pFile);
  printf("Total Width : %E\n Created File : %s", res, Filename);

  // Free allocated memory
  free(wE1);
  free(xE1);
  free(wE2);
  free(xE2);
  free(wct1);
  free(xct1);
  free(wct2);
  free(xct2);
  free(wct3);
  free(xct3);
  free(wp2);
  free(xp2);
  free(wp3);
  free(xp3);

  hipFree(d_wE1);
  hipFree(d_xE1);
  hipFree(d_wE2);
  hipFree(d_xE2);
  hipFree(d_wct1);
  hipFree(d_xct1);
  hipFree(d_wct2);
  hipFree(d_xct2);
  hipFree(d_wct3);
  hipFree(d_xct3);
  hipFree(d_wp2);
  hipFree(d_xp2);
  hipFree(d_wp3);
  hipFree(d_xp3);
}

// Estimate the runtime of the given choice of number of nodes
void EstimateRuntime() {
  double i = (5580000.0 * sett.orderE1 * sett.orderE2 * sett.orderct1 *
              sett.orderct2 * sett.orderct3 * sett.orderp2 * sett.orderp3) /
             34359738368;
  printf("%ih%im%is%ims\n", (int)(i / (60 * 60 * 1000)),
         (int)(i / (1000 * 60)) % 60, (int)(i / 1000) % 60, ((int)i) % 1000);
}

// Initialise fixed parameters on the device
void InitParameters() {
  hipMemcpyToSymbol(HIP_SYMBOL(m), &sett.m, sizeof(double), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(me), &sett.me, sizeof(double), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(m2), &sett.m2, sizeof(double), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Gf), &sett.Gf, sizeof(double), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(g), &sett.g, sizeof(double), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderE1), &sett.orderE1, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderE2), &sett.orderE2, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderct1), &sett.orderct1, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderct2), &sett.orderct2, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderct3), &sett.orderct3, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderp2), &sett.orderp2, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(orderp3), &sett.orderp3, sizeof(int), 0,
                     hipMemcpyHostToDevice);
}

// Read space-seperated field of a line
const char *getfield(char *line, int num) {
  const char *tok;
  for (tok = strtok(strdup(line), "	"); tok && *tok;
       tok = strtok(NULL, "	\n")) {
    if (!--num)
      return tok;
  }
  return NULL;
}

// Parse char* line to settings se
void ReadSettings(char *line, struct Settings &se) {
  se.m2 = atof(getfield(line, 1));
  se.orderE1 = atoi(getfield(line, 2));
  se.orderE2 = atoi(getfield(line, 3));
  se.orderct1 = atoi(getfield(line, 4));
  se.orderct2 = atoi(getfield(line, 5));
  se.orderct3 = atoi(getfield(line, 6));
  se.orderp2 = atoi(getfield(line, 7));
  se.orderp3 = atoi(getfield(line, 8));
  se.done = atoi(getfield(line, 9));
  if (se.done)
    se.filename = string(getfield(line, 10));
}

// Read schedule from external file to avoid recompiling
void ReadSchedule(vector<Settings> &listSett) {
  listSett.clear();
  FILE *stream = fopen("schedule.txt", "r");
  char line[1024];
  while (fgets(line, 1024, stream)) {

    char *tmp = strdup(line);
    struct Settings se;
    ReadSettings(tmp, se);
    listSett.push_back(se);
    free(tmp);
  }
  fclose(stream);
}

// Write new schedule-status to file
void WriteSchedule(vector<Settings> listSett) {
  FILE *stream = fopen("schedule.txt", "w");
  for (int i = 0; i < static_cast<int>(listSett.size()); i++) {
    Settings curr = listSett[i];
    if (curr.done)
      fprintf(stream,
              "%f	%i	%i	%i	%i	%i	%i	"
              "%i	%i	%s\n",
              curr.m2, curr.orderE1, curr.orderE2, curr.orderct1, curr.orderct2,
              curr.orderct3, curr.orderp2, curr.orderp3, curr.done,
              curr.filename.c_str());
    else
      fprintf(
          stream,
          "%f	%i	%i	%i	%i	%i	%i	%i	%i\n",
          curr.m2, curr.orderE1, curr.orderE2, curr.orderct1, curr.orderct2,
          curr.orderct3, curr.orderp2, curr.orderp3, curr.done);
  }
  fclose(stream);
}

// Check for new work and load settings
int FetchNewWork(vector<Settings> &listSett) {
  ReadSchedule(listSett);
  for (int i = 0; i < static_cast<int>(listSett.size()); i++) {
    if (!listSett[i].done) {
      sett = listSett[i];
      printf("\n Found new Work\n");
      return 1;
    }
  }
  return 0;
}

// Runs integrations as long as new work is scheduled
void RunScheduledJobs() {
  srand(time(NULL));
  vector<Settings> listSett;
  while (FetchNewWork(listSett)) {
    InitParameters();
    EstimateRuntime();
    string filenam = "Results/Scalar/" + to_string(rand() % 100000) + ".csv";
    IntegrateOnGPU(filenam.c_str());
    ReadSchedule(listSett);
    for (int i = 0; i < static_cast<int>(listSett.size()); i++) {

      if (sett == listSett[i]) {
        listSett[i].done = 1;
        listSett[i].filename = filenam;
        break;
      }
    }
    WriteSchedule(listSett);
  }
}

// Serves as a quick test for debug-purposes
void Quicktest() {
  sett.orderE1 = 10;
  sett.orderE2 = 10;
  sett.orderct1 = 10;
  sett.orderct2 = 10;
  sett.orderct3 = 10;
  sett.orderp2 = 10;
  sett.orderp3 = 10;
  sett.m2 = 0.01;
  EstimateRuntime();
  InitParameters();
  IntegrateOnGPU("/tmp/test.txt");
}

int main(int argc, char *argv[]) {

  Quicktest();
  return 0;
}
